#include "hip/hip_runtime.h"
/**
 * @author Alexander Van Craen
 * @author Marcel Breyer
 * @copyright 2018-today The PLSSVM project - All Rights Reserved
 * @license This file is part of the PLSSVM project which is released under the MIT license.
 *          See the LICENSE.md file in the project root for full license information.
 */

#include "plssvm/backends/CUDA/csvm.hpp"

#include "plssvm/backends/CUDA/detail/device_ptr.cuh"  // plssvm::cuda::detail::device_ptr
#include "plssvm/backends/CUDA/detail/utility.cuh"     // plssvm::cuda::detail::device_synchronize, plssvm::detail::cuda::get_device_count, plssvm::detail::cuda::set_device, plssvm::detail::cuda::peek_at_last_error
#include "plssvm/backends/CUDA/exceptions.hpp"         // plssvm::cuda::backend_exception
#include "plssvm/backends/CUDA/predict_kernel.cuh"     // plssvm::cuda::kernel_w, plssvm::cuda::predict_points_poly, plssvm::cuda::predict_points_rbf
#include "plssvm/backends/CUDA/q_kernel.cuh"           // plssvm::cuda::device_kernel_q_linear, plssvm::cuda::device_kernel_q_poly, plssvm::cuda::device_kernel_q_radial
#include "plssvm/backends/CUDA/svm_kernel.cuh"         // plssvm::cuda::device_kernel_linear, plssvm::cuda::device_kernel_poly, plssvm::cuda::device_kernel_radial
#include "plssvm/backends/gpu_csvm.hpp"                // plssvm::detail::gpu_csvm
#include "plssvm/detail/assert.hpp"                    // PLSSVM_ASSERT
#include "plssvm/detail/execution_range.hpp"           // plssvm::detail::execution_range
#include "plssvm/exceptions/exceptions.hpp"            // plssvm::exception
#include "plssvm/kernel_types.hpp"                     // plssvm::kernel_type
#include "plssvm/parameter.hpp"                        // plssvm::parameter
#include "plssvm/target_platforms.hpp"                 // plssvm::target_platform

#include "plssvm/backends/CUDA/transform_kernel.cuh"         // plssvm::cuda::device_kernel_linear, plssvm::cuda::device_kernel_poly, plssvm::cuda::device_kernel_radial

#include "fmt/core.h"     // fmt::print, fmt::format
#include "fmt/ostream.h"  // can use fmt using operator<< overloads

#include <exception>  // std::terminate
#include <numeric>    // std::iota
#include <utility>    // std::pair, std::make_pair
#include <vector>     // std::vector

namespace plssvm::cuda {

csvm::csvm(const parameter &params) :
    gpu_csvm::gpu_csvm{ params } {
    // check if supported target platform has been selected
    if (target_ != target_platform::automatic && target_ != target_platform::gpu_nvidia) {
        throw backend_exception{ fmt::format("Invalid target platform '{}' for the CUDA backend!", target_) };
    } else {
#if !defined(PLSSVM_HAS_NVIDIA_TARGET)
        throw backend_exception{ fmt::format("Requested target platform {} that hasn't been enabled using PLSSVM_TARGET_PLATFORMS!", target_) };
#endif
    }

    if (print_info_) {
        fmt::print("Using CUDA as backend.\n");
    }

    // get all available devices wrt the requested target platform
    devices_.resize(std::min<std::size_t>(detail::get_device_count(), num_features_));
    std::iota(devices_.begin(), devices_.end(), 0);

    // throw exception if no CUDA devices could be found
    if (devices_.empty()) {
        throw backend_exception{ "CUDA backend selected but no CUDA devices were found!" };
    }

    // polynomial and rbf kernel currently only support single GPU execution
    if (kernel_ == kernel_type::polynomial || kernel_ == kernel_type::rbf) {
        devices_.resize(1);
    }

    // For tests running on single gpu, resize to one
    #if defined(SINGLE_TEST)
        devices_.resize(1);
    #endif

    // resize vectors accordingly
    data_d_.resize(devices_.size());
    data_d_f_.resize(devices_.size());
    data_last_d_.resize(devices_.size());
    data_last_d_f_.resize(devices_.size());

    if (print_info_) {
        // print found CUDA devices
        fmt::print("Found {} CUDA device(s):\n", devices_.size());
        for (typename std::vector<queue_type>::size_type device = 0; device < devices_.size(); ++device) {
            hipDeviceProp_t prop{};
            hipGetDeviceProperties(&prop, devices_[device]);
            fmt::print("  [{}, {}, {}.{}]\n", devices_[device], prop.name, prop.major, prop.minor);
        }
        fmt::print("\n");
    }
}


csvm::~csvm() {
    try {
        // be sure that all operations on the CUDA devices have finished before destruction
        for (const queue_type &device : devices_) {
            detail::device_synchronize(device);
        }
    } catch (const plssvm::exception &e) {
        fmt::print("{}\n", e.what_with_loc());
        std::terminate();
    }
}

void csvm::device_synchronize(queue_type &queue) {
    detail::device_synchronize(queue);
}

std::pair<dim3, dim3> execution_range_to_native(const ::plssvm::detail::execution_range &range) {
    dim3 grid(range.grid[0], range.grid[1], range.grid[2]);
    dim3 block(range.block[0], range.block[1], range.block[2]);
    return std::make_pair(grid, block);
}


void csvm::run_q_kernel(const std::size_t device, const ::plssvm::detail::execution_range &range, device_ptr_type &q_d, const std::size_t num_features) {
    auto [grid, block] = execution_range_to_native(range);

    detail::set_device(device);
    switch (kernel_) {
        case kernel_type::linear:
            cuda::device_kernel_q_linear<<<grid, block>>>(q_d.get(), data_d_[device].get(), data_last_d_[device].get(), num_rows_, num_features, gamma_);
            break;
        case kernel_type::polynomial:
            PLSSVM_ASSERT(device == 0, "The polynomial kernel function currently only supports single GPU execution!");
            cuda::device_kernel_q_poly<<<grid, block>>>(q_d.get(), data_d_[device].get(), data_last_d_[device].get(), num_rows_, num_cols_, degree_, gamma_, coef0_);
            break;
        case kernel_type::rbf:
            PLSSVM_ASSERT(device == 0, "The radial basis function kernel function currently only supports single GPU execution!");
            cuda::device_kernel_q_radial<<<grid, block>>>(q_d.get(), data_d_[device].get(), data_last_d_[device].get(), num_rows_, num_cols_, gamma_);
            break;
    }
    detail::peek_at_last_error();
}

void csvm::run_q_kernel_f(const std::size_t device, const ::plssvm::detail::execution_range &range, device_ptr_type_float &q_d_f, const std::size_t num_features) {
    auto [grid, block] = execution_range_to_native(range);

    detail::set_device(device);
    switch (kernel_) {
        case kernel_type::linear:
            cuda::device_kernel_q_linear_f<<<grid, block>>>(q_d_f.get(), data_d_f_[device].get(), data_last_d_f_[device].get(), num_rows_, num_features, gamma_f_);
            break;
        case kernel_type::polynomial:
            PLSSVM_ASSERT(device == 0, "The polynomial kernel function currently only supports single GPU execution!");
            cuda::device_kernel_q_poly_f<<<grid, block>>>(q_d_f.get(), data_d_f_[device].get(), data_last_d_f_[device].get(), num_rows_, num_cols_, degree_, gamma_f_, coef0_);
            break;
        case kernel_type::rbf:
            PLSSVM_ASSERT(device == 0, "The radial basis function kernel function currently only supports single GPU execution!");
            cuda::device_kernel_q_radial_f<<<grid, block>>>(q_d_f.get(), data_d_f_[device].get(), data_last_d_f_[device].get(), num_rows_, num_cols_, gamma_f_);
            break;
    }
    detail::peek_at_last_error();
}

void csvm::run_svm_kernel(const std::size_t device, const ::plssvm::detail::execution_range &range, const device_ptr_type &q_d, device_ptr_type &r_d, const device_ptr_type &x_d, const real_type add, const std::size_t num_features) {
    auto [grid, block] = execution_range_to_native(range);

    detail::set_device(device);
    switch (kernel_) {
        case kernel_type::linear:
            // PLSSVM_CUDA_ERROR_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(cuda::device_kernel_linear<double>), hipFuncAttributeMaxDynamicSharedMemorySize, INTERNAL_BLOCK_SIZE * INTERNAL_BLOCK_SIZE * (256 + 2) * 8 ));
            cuda::device_kernel_linear<<<grid, block>>>(q_d.get(), r_d.get(), x_d.get(), data_d_[device].get(), QA_cost_, 1 / cost_, num_rows_, num_features, add, gamma_, device);
            break;
        case kernel_type::polynomial:
            PLSSVM_ASSERT(device == 0, "The polynomial kernel function currently only supports single GPU execution!");
            cuda::device_kernel_poly<<<grid, block>>>(q_d.get(), r_d.get(), x_d.get(), data_d_[device].get(), QA_cost_, 1 / cost_, num_rows_, num_cols_, add, degree_, gamma_, coef0_);
            break;
        case kernel_type::rbf:
            PLSSVM_ASSERT(device == 0, "The radial basis function kernel function currently only supports single GPU execution!");
            cuda::device_kernel_radial<<<grid, block>>>(q_d.get(), r_d.get(), x_d.get(), data_d_[device].get(), QA_cost_, 1 / cost_, num_rows_, num_cols_, add, gamma_);
            break;
    }
    detail::peek_at_last_error();
}

void csvm::run_svm_kernel_td(const std::size_t device, const ::plssvm::detail::execution_range &range, const device_ptr_type &q_d, device_ptr_type &r_d, const device_ptr_type &x_d, const real_type add, const std::size_t num_features) {
    auto [grid, block] = execution_range_to_native(range);

    size_t dyn_sha_mem = ((BLOCK_SIZE + 4) * BLOCK_OFF) * sizeof(double); 
    // fmt::print("grid 0 1 2: {} {} {} - block: {} {} {} \n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

    detail::set_device(device);
    switch (kernel_) {
        case kernel_type::linear:             
            PLSSVM_CUDA_ERROR_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(cuda::device_kernel_linear_td), hipFuncAttributeMaxDynamicSharedMemorySize, dyn_sha_mem));
            cuda::device_kernel_linear_td<<<grid, block, dyn_sha_mem>>>(q_d.get(), r_d.get(), x_d.get(), data_d_[device].get(), QA_cost_, 1 / cost_, num_rows_, num_features, add, gamma_, device); // , INTERNAL_BLOCK_SIZE * INTERNAL_BLOCK_SIZE * (256 + 2) * 8
            break;
        case kernel_type::polynomial:
            PLSSVM_ASSERT(device == 0, "The polynomial kernel function currently only supports single GPU execution!");
            PLSSVM_CUDA_ERROR_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(cuda::device_kernel_poly_td), hipFuncAttributeMaxDynamicSharedMemorySize, dyn_sha_mem));
            cuda::device_kernel_poly_td<<<grid, block, dyn_sha_mem>>>(q_d.get(), r_d.get(), x_d.get(), data_d_[device].get(), QA_cost_, 1 / cost_, num_rows_, num_cols_, add, degree_, gamma_, coef0_);
            break;
        case kernel_type::rbf:
            throw backend_exception{ fmt::format("Radial Kernel not usable with TENSOR, use standard-Kernel!")};             
            break;
    }
    detail::peek_at_last_error();
    // fmt::print("Hi after Kernel \n");
}

void csvm::run_svm_kernel_tf(const std::size_t device, const ::plssvm::detail::execution_range &range, const device_ptr_type_float &q_d, device_ptr_type_float &r_d, const device_ptr_type_float &x_d, const float add, const std::size_t num_features) {
    auto [grid, block] = execution_range_to_native(range);

    // fmt::print("grid 0 1 2: {} {} {} - block: {} {} {} \n", grid.x, grid.y, grid.z, block.x, block.y, block.z);

    size_t dyn_sha_mem = ((BLOCK_SIZE_F + 4) * BLOCK_OFF_F)*sizeof(float); //Matrix, Ausgangsmatrix i und j + Vec

    detail::set_device(device);
    switch (kernel_) {
        case kernel_type::linear:             
            PLSSVM_CUDA_ERROR_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(cuda::device_kernel_linear_tf), hipFuncAttributeMaxDynamicSharedMemorySize, dyn_sha_mem));
            cuda::device_kernel_linear_tf<<<grid, block, dyn_sha_mem>>>(q_d.get(), r_d.get(), x_d.get(), data_d_f_[device].get(), QA_cost_f_, 1 / cost_f_, num_rows_, num_features, add, gamma_f_, device); // , INTERNAL_BLOCK_SIZE * INTERNAL_BLOCK_SIZE * (256 + 2) * 8
            break;
        case kernel_type::polynomial:
            PLSSVM_ASSERT(device == 0, "The polynomial kernel function currently only supports single GPU execution!");
            PLSSVM_CUDA_ERROR_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(cuda::device_kernel_poly_tf), hipFuncAttributeMaxDynamicSharedMemorySize, dyn_sha_mem));
            cuda::device_kernel_poly_tf<<<grid, block, dyn_sha_mem>>>(q_d.get(), r_d.get(), x_d.get(), data_d_f_[device].get(), QA_cost_f_, 1 / cost_f_, num_rows_, num_cols_, add, degree_, gamma_f_, coef0_f_);
            break;
        case kernel_type::rbf:
            throw backend_exception{ fmt::format("Radial Kernel not usable with TENSOR, use standard-Kernel!")};
            break;
    }
    detail::peek_at_last_error();
}

void csvm::run_svm_kernel_f(const std::size_t device, const ::plssvm::detail::execution_range &range, const device_ptr_type_float &q_d, device_ptr_type_float &r_d, const device_ptr_type_float &x_d, const float add, const std::size_t num_features) {
    auto [grid, block] = execution_range_to_native(range);

    detail::set_device(device);
    switch (kernel_) {
        case kernel_type::linear:
            cuda::device_kernel_linear<<<grid, block>>>(q_d.get(), r_d.get(), x_d.get(), data_d_f_[device].get(), QA_cost_f_, 1 / cost_f_, num_rows_, num_features, add, gamma_f_, device);
            break;
        case kernel_type::polynomial:
            PLSSVM_ASSERT(device == 0, "The polynomial kernel function currently only supports single GPU execution!");
            cuda::device_kernel_poly<<<grid, block>>>(q_d.get(), r_d.get(), x_d.get(), data_d_f_[device].get(), QA_cost_f_, 1 / cost_f_, num_rows_, num_cols_, add, degree_, gamma_f_, coef0_f_);
            break;
        case kernel_type::rbf:
            PLSSVM_ASSERT(device == 0, "The radial basis function kernel function currently only supports single GPU execution!");
            cuda::device_kernel_radial<<<grid, block>>>(q_d.get(), r_d.get(), x_d.get(), data_d_f_[device].get(), QA_cost_f_, 1 / cost_f_, num_rows_, num_cols_, add, gamma_f_);
            break;
    }
    detail::peek_at_last_error();
}

void csvm::run_w_kernel(const std::size_t device, const ::plssvm::detail::execution_range &range, device_ptr_type &w_d, const device_ptr_type &alpha_d, const std::size_t num_features) {
    auto [grid, block] = execution_range_to_native(range);

    detail::set_device(device);
    cuda::device_kernel_w_linear<<<grid, block>>>(w_d.get(), data_d_[device].get(), data_last_d_[device].get(), alpha_d.get(), num_data_points_, num_features);
    detail::peek_at_last_error();
}

void csvm::run_predict_kernel(const ::plssvm::detail::execution_range &range, device_ptr_type &out_d, const device_ptr_type &alpha_d, const device_ptr_type &point_d, const std::size_t num_predict_points) {
    auto [grid, block] = execution_range_to_native(range);

    detail::set_device(0);
    switch (kernel_) {
        case kernel_type::linear:
            break;
        case kernel_type::polynomial:
            cuda::device_kernel_predict_poly<<<grid, block>>>(out_d.get(), data_d_[0].get(), data_last_d_[0].get(), alpha_d.get(), num_data_points_, point_d.get(), num_predict_points, num_features_, degree_, gamma_, coef0_);
            break;
        case kernel_type::rbf:
            cuda::device_kernel_predict_radial<<<grid, block>>>(out_d.get(), data_d_[0].get(), data_last_d_[0].get(), alpha_d.get(), num_data_points_, point_d.get(), num_predict_points, num_features_, gamma_);
            break;
    }
    detail::peek_at_last_error();
}

void csvm::run_transformation_kernel_df(const std::size_t device, const ::plssvm::detail::execution_range &range, device_ptr_type_float &float_out, const device_ptr_type &double_in) {
    auto [grid, block] = execution_range_to_native(range);

    detail::set_device(device);
    cuda::device_kernel_cast_double_to_float<<<grid, block>>>(double_in.get(), float_out.get(), double_in.size());
    detail::peek_at_last_error();
}

void csvm::run_transformation_kernel_fd(const std::size_t device, const ::plssvm::detail::execution_range &range, device_ptr_type &double_out, const device_ptr_type_float &float_in) {
    auto [grid, block] = execution_range_to_native(range);

    detail::set_device(device);
    cuda::device_kernel_cast_float_to_double<<<grid, block>>>(float_in.get(), double_out.get(), float_in.size());
    detail::peek_at_last_error();
}

//template class csvm<double>;

}  // namespace plssvm::cuda
